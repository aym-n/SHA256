#include <hip/hip_runtime.h>

#include <chrono>
#include <cstring>
#include <iomanip>
#include <iostream>
#include <vector>

using namespace std;
__constant__ uint32_t K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5, 0xd807aa98, 0x12835b01, 0x243185be,
    0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174, 0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa,
    0x5cb0a9dc, 0x76f988da, 0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967, 0x27b70a85,
    0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85, 0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070, 0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f,
    0x682e6ff3, 0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

__constant__ uint8_t constant_input[64];

__device__ uint32_t rotr(uint32_t x, int n) { return (x >> n) | (x << (32 - n)); }
__device__ uint32_t ch(uint32_t x, uint32_t y, uint32_t z) { return (x & y) ^ (~x & z); }
__device__ uint32_t maj(uint32_t x, uint32_t y, uint32_t z) { return (x & y) ^ (x & z) ^ (y & z); }
__device__ uint32_t sigma0(uint32_t x) { return rotr(x, 2) ^ rotr(x, 13) ^ rotr(x, 22); }
__device__ uint32_t sigma1(uint32_t x) { return rotr(x, 6) ^ rotr(x, 11) ^ rotr(x, 25); }
__device__ uint32_t gamma0(uint32_t x) { return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3); }
__device__ uint32_t gamma1(uint32_t x) { return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10); }

__global__ void sha256_kernel(uint32_t *output, size_t num_hashes)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_hashes)
    return;

  uint32_t state[8] = {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};
  uint32_t w[64];

#pragma unroll
  for (int i = 0; i < 16; ++i)
  {
    w[i] =
        (constant_input[i * 4] << 24) | (constant_input[i * 4 + 1] << 16) | (constant_input[i * 4 + 2] << 8) | (constant_input[i * 4 + 3]);
  }

#pragma unroll
  for (int i = 16; i < 64; ++i) w[i] = gamma1(w[i - 2]) + w[i - 7] + gamma0(w[i - 15]) + w[i - 16];

  uint32_t a = state[0], b = state[1], c = state[2], d = state[3];
  uint32_t e = state[4], f = state[5], g = state[6], h = state[7];

#pragma unroll
  for (int i = 0; i < 64; ++i)
  {
    uint32_t t1 = h + sigma1(e) + ch(e, f, g) + K[i] + w[i];
    uint32_t t2 = sigma0(a) + maj(a, b, c);
    h = g;
    g = f;
    f = e;
    e = d + t1;
    d = c;
    c = b;
    b = a;
    a = t1 + t2;
  }

  state[0] += a;
  state[1] += b;
  state[2] += c;
  state[3] += d;
  state[4] += e;
  state[5] += f;
  state[6] += g;
  state[7] += h;

  for (int i = 0; i < 8; ++i) output[idx * 8 + i] = state[i];
}

void benchmark(const string &input, int num_hashes)
{
  cout << "[Starting Benchmark]" << endl;
  size_t len = input.size();

  uint8_t h_input[64] = {0};
  memcpy(h_input, input.data(), len);
  h_input[len] = 0x80;  // Padding
                        //
  uint64_t bit_len = len * 8;
  for (int i = 0; i < 8; ++i) h_input[56 + i] = (bit_len >> (56 - 8 * i)) & 0xff;

  hipMemcpyToSymbol(HIP_SYMBOL(constant_input), h_input, 64);
  vector<uint32_t> h_output(num_hashes * 8);
  uint32_t *d_output;
  hipMalloc(&d_output, num_hashes * 8 * sizeof(uint32_t));

  int threadsPerBlock = 256;
  int blocksPerGrid = (num_hashes + threadsPerBlock - 1) / threadsPerBlock;

  cout << "[Start Calculating]" << endl;
  auto start = chrono::high_resolution_clock::now();
  sha256_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_output, num_hashes);
  hipDeviceSynchronize();
  auto end = chrono::high_resolution_clock::now();

  hipMemcpy(h_output.data(), d_output, num_hashes * 8 * sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipFree(d_output);

  auto duration = chrono::duration_cast<chrono::milliseconds>(end - start).count();
  cout << "Time taken: " << duration << " ms\n";
  cout << "Speed: " << (float)num_hashes / (duration / 1000.0) / 1e6 << " MH/s\n";

  cout << "\nSHA256:\n";
  for (int i = 0; i < 8; ++i) cout << hex << setw(8) << setfill('0') << h_output[i] << " ";
  cout << dec << endl;
}

int main()
{
  string input = "Hello Vicharak";
  benchmark(input, 10000000);
  return 0;
}
